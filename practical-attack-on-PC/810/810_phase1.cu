#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

#include <ctime>
#include <chrono>

using namespace std;
#define TYPE unsigned int
typedef signed char u8;
#define BASE_CUDA_CHECK(condition) { GPUAssert((condition), __FILE__, __LINE__); }
typedef unsigned int u32;



__device__ void Trivium_device(u32* keystream, u32* iv, u32 key_index)
{

	u32 roundnum = 810;

	//the original secret variables  
	u32 key[10] = { 41,35,190,132,225,108,214,174,82,144 };

	u32 z1 = 0;
	u32 roundnum_word = 0;
	u32 roundnum_bit = 0;
	u32 t1, t2, t3, i = 0;
	u32 s0, s1, s2, s3, s4, s5, s6, s7, s8, s9 = 0;
	u32 temp1, temp2, temp3, temp4, temp5, temp6 = 0;
	u32 z = 0;

	s0 = key[0] ^ (key[1] << 8) ^ (key[2] << 16) ^ (key[3] << 24);  //32k


	s1 = key[4] ^ (key[5] << 8) ^ (key[6] << 16) ^ (key[7] << 24);  //32k


	s2 = key[8] ^ (key[9] << 8); 


	s3 = iv[0];

	s4 = iv[1];

	s5 = iv[2];


	s6 = 0;


	s7 = 0;


	s8 = 0;


	s9 = 0x00007000;  //7000  =  0111 + 12*0



	//65         92               161          176        242           287
	//s2[30]     s2[3]           s5[27]       s5[12]      s8[30]       s9[17]



	//64         91               160          175        241           286
	//s2[31]     s2[4]           s5[28]       s5[13]      s8[31]       s9[18]




	roundnum_word = roundnum / 32;
	roundnum_bit = roundnum % 32;
	for (i = 0;i < roundnum_word;i++)
	{

		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;//((s2<<30)|(s1>>2))^((s2<<3)|(s1>>29))^(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);

		t2 = temp3 ^ temp4;//((s5<<27)|(s4>>5))^((s5<<12)|(s4>>20))^((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19))^((s8<<9)|(s7>>23));

		t3 = temp5 ^ temp6;//((s8<<30)|(s7>>2))^((s9<<17)| (s8>>15))^

		//z = t1^t2^t3;

	//	t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);//(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//	t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);//(((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19)))^((s8<<9)|(s7>>23));
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//	t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);//(()&())^()
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;




		// update register 1
		s2 = (s1) & (0x1FFFFFFF);  //29λ
		s1 = s0;
		s0 = t3;

		//	update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//	update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}
	if (roundnum_bit != 0)
	{
		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;
		t2 = temp3 ^ temp4;
		t3 = temp5 ^ temp6;
		z1 = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;
		//z1=z;
		//t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;

		// update register 1
		s2 = (s1) & (0x1FFFFFFF);
		s1 = s0;
		s0 = t3;

		//update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}

	temp1 = (s2 << 30) | (s1 >> 2);
	temp2 = (s2 << 3) | (s1 >> 29);
	temp3 = (s5 << 27) | (s4 >> 5);
	temp4 = (s5 << 12) | (s4 >> 20);
	temp5 = (s8 << 30) | (s7 >> 2);
	temp6 = (s9 << 17) | (s8 >> 15);



	int outbit = 0;
	z = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;

	if (roundnum_bit != 0)
		outbit = (z1 >> (31 - roundnum_bit)) & 0x1;
	else
		outbit = z >> 31;

	keystream[0] = outbit;
}

__global__ void global_multi_thread(u32* part_term_dev_divide, u32 loop) {
	//u32 loadkey[10] = { 41,35,190,132,225,108,214,174,82,144 };

	u32 tid_16, i;
	u32 keystream[1];
	u32 offset_28;
	u32 streambit = 0;
	u32 tmp;

	//0-2^16
	//0, 2, 4, 6, 8, 10, 11, 12, 14, 15, 17, 19, 20, 21, 22, 23
	tid_16 = loop;
	tmp = (threadIdx.x + blockIdx.x * blockDim.x);




	for (i = 0; i < 32; i++) {
		offset_28 = ((tmp << 5) | i);


		u32 cube[3] = { 0 };


		cube[0] = (tid_16 & 0x1) | (((tid_16 >> 1) & 0x1) << 2) | (((tid_16 >> 2) & 0x1) << 4) | (((tid_16 >> 3) & 0x1) << 6) | (((tid_16 >> 4) & 0x1) << 8) | (((tid_16 >> 5) & 0x1) << 10) | (((tid_16 >> 6) & 0x1) << 11) | (((tid_16 >> 7) & 0x1) << 12) | (((tid_16 >> 8) & 0x1) << 14) | (((tid_16 >> 9) & 0x1) << 15) | (((tid_16 >> 10) & 0x1) << 17) | (((tid_16 >> 11) & 0x1) << 19) | (((tid_16 >> 12) & 0x1) << 20) | (((tid_16 >> 13) & 0x1) << 21) | (((tid_16 >> 14) & 0x1) << 22) | (((tid_16 >> 15) & 0x1) << 23) | ((offset_28 & 0x1) << 25) | (((offset_28 >> 1) & 0x1) << 27) | (((offset_28 >> 2) & 0x1) << 29) | (((offset_28 >> 3) & 0x1) << 30);

		cube[1] = (((offset_28 >> 4) & 0x1)) | (((offset_28 >> 5) & 0x1) << 2) | (((offset_28 >> 6) & 0x1) << 4) | (((offset_28 >> 7) & 0x1) << 5) | (((offset_28 >> 8) & 0x1) << 7) | (((offset_28 >> 9) & 0x1) << 9) | (((offset_28 >> 10) & 0x1) << 11) | (((offset_28 >> 11) & 0x1) << 13) | (((offset_28 >> 12) & 0x1) << 15) | (((offset_28 >> 13) & 0x1) << 16) | (((offset_28 >> 14) & 0x1) << 18) | (((offset_28 >> 15) & 0x1) << 20) | (((offset_28 >> 16) & 0x1) << 21) | (((offset_28 >> 17) & 0x1) << 22) | (((offset_28 >> 18) & 0x1) << 23) | (((offset_28 >> 19) & 0x1) << 25) | (((offset_28 >> 20) & 0x1) << 28) | (((offset_28 >> 21) & 0x1) << 30);

		cube[2] = (((offset_28 >> 22) & 0x1)) | (((offset_28 >> 23) & 0x1) << 3) | (((offset_28 >> 24) & 0x1) << 6) | (((offset_28 >> 25) & 0x1) << 8) | (((offset_28 >> 26) & 0x1) << 11) | (((offset_28 >> 27) & 0x1) << 15);

		Trivium_device(keystream, cube, i);
		streambit |= ((keystream[0] & 0x01) << (i));



	}

	/*printf("%u", streambit);*/

	part_term_dev_divide[threadIdx.x + blockIdx.x * blockDim.x] = streambit;



	__syncthreads();
}


__global__ void  getsum(u32* part_term_host_divide, u32 loop, u32* sum1_dev, u32 dim) {
	u32 A = 0;
	u32 A1 = 0;
	u32 B = 0;
	u32 B_5 = 0;
	u32 index = threadIdx.x + blockIdx.x * blockDim.x;

	sum1_dev[index] = 0;

	for (A1 = dim * index; A1 < dim * (index + 1); A1++) {

		for (B_5 = 0; B_5 < 32; B_5++) {

			B = ((A1 << 5) | B_5);
			A = loop;

			if ((((B >> 11) & 0x1) == 0) && (((B >> 2) & 0x1) == 0) && (((A >> 8) & 0x1) == 0) && (((A >> 1) & 0x1) == 0)) {
				sum1_dev[index] ^= (((part_term_host_divide[A1]) >> (B_5)) & 0X1);
			}
			if ((((B >> 15) & 0x1) == 0) && (((B >> 2) & 0x1) == 0) && (((A >> 14) & 0x1) == 0) && (((A >> 11) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 1);
			}

			if ((((B >> 23) & 0x1) == 0) && (((B >> 16) & 0x1) == 0) && (((B >> 12) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 2);
			}

			if ((((A >> 11) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 3);
			}

			if ((((B >> 17) & 0x1) == 0) && (((B >> 15) & 0x1) == 0) && (((B >> 5) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 4);
			}
			if ((((B >> 19) & 0x1) == 0) && (((B >> 14) & 0x1) == 0) && (((A >> 1) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 5);
			}

			if ((((B >> 3) & 0x1) == 0) && (((A >> 9) & 0x1) == 0) && (((A >> 6) & 0x1) == 0) && (((A >> 1) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 6);
			}

			if ((((B >> 19) & 0x1) == 0) && (((B >> 3) & 0x1) == 0) && (((A >> 9) & 0x1) == 0) && (((A >> 7) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 7);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 12) & 0x1) == 0) && (((B >> 9) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 8);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 19) & 0x1) == 0) && (((B >> 13) & 0x1) == 0) && (((A >> 9) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 9);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 17) & 0x1) == 0) && (((B >> 16) & 0x1) == 0) && (((B >> 2) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 10);
			}
			if ((((B >> 24) & 0x1) == 0) && (((B >> 23) & 0x1) == 0) && (((B >> 13) & 0x1) == 0) && (((B >> 2) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 11);
			}
			if ((((B >> 12) & 0x1) == 0) && (((B >> 10) & 0x1) == 0) && (((B >> 1) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 12);
			}
			if ((((B >> 18) & 0x1) == 0) && (((B >> 12) & 0x1) == 0) && (((A >> 15) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 13);
			}
			if ((((B >> 27) & 0x1) == 0) && (((B >> 17) & 0x1) == 0) && (((B >> 7) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 14);
			}
			if ((((B >> 21) & 0x1) == 0) && (((A >> 12) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 15);
			}
			if ((((B >> 13) & 0x1) == 0) && (((B >> 21) & 0x1) == 0) && (((B >> 5) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 16);
			}
			if ((((B >> 20) & 0x1) == 0) && (((B >> 19) & 0x1) == 0) && (((A >> 8) & 0x1) == 0) && (((A >> 1) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 17);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 3) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 18);
			}
			if ((((B >> 18) & 0x1) == 0) && (((B >> 2) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 19);
			}
			if ((((B >> 22) & 0x1) == 0) && (((B >> 20) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 20);
			}
			if ((((B >> 8) & 0x1) == 0) && (((B >> 3) & 0x1) == 0) && (((A >> 9) & 0x1) == 0) && (((A >> 7) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 21);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 7) & 0x1) == 0) && (((B >> 5) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 22);
			}
			if ((((B >> 17) & 0x1) == 0) && (((A >> 6) & 0x1) == 0) && (((A >> 2) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 23);
			}
			if ((((B >> 22) & 0x1) == 0) && (((B >> 18) & 0x1) == 0) && (((B >> 6) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 24);
			}
			if ((((B >> 20) & 0x1) == 0) && (((B >> 17) & 0x1) == 0) && (((A >> 11) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 25);
			}
			if ((((B >> 17) & 0x1) == 0) && (((B >> 13) & 0x1) == 0) && (((A >> 10) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 26);
			}
			if ((((B >> 27) & 0x1) == 0) && (((B >> 12) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 27);
			}
			if ((((B >> 21) & 0x1) == 0) && (((B >> 0) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 28);
			}
			if ((((B >> 23) & 0x1) == 0) && (((B >> 21) & 0x1) == 0) && (((B >> 5) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 29);
			}
			if ((((B >> 26) & 0x1) == 0) && (((B >> 5) & 0x1) == 0) && (((B >> 2) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 30);
			}
			if ((((B >> 18) & 0x1) == 0) && (((B >> 15) & 0x1) == 0) && (((B >> 5) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 31);
			}




		}


	}

	__syncthreads();
}

__global__ void  getsum2(u32* part_term_host_divide, u32 loop, u32* sum1_dev, u32 dim) {
	u32 A = 0;
	u32 A1 = 0;
	u32 B = 0;
	u32 B_5 = 0;
	u32 index = threadIdx.x + blockIdx.x * blockDim.x;

	sum1_dev[index] = 0;
	for (A1 = dim * index; A1 < dim * (index + 1); A1++) {



		for (B_5 = 0; B_5 < 32; B_5++) {

			B = ((A1 << 5) | B_5);

			A = loop;

			if ((((B >> 12) & 0x1) == 0) && (((B >> 0) & 0x1) == 0) && (((A >> 15) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1));

			}

			if ((((A >> 6) & 0x1) == 0) && (((A >> 2) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 1);

			}

			if ((((B >> 21) & 0x1) == 0) && (((B >> 13) & 0x1) == 0) && (((B >> 5) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 2);

			}

			if ((((B >> 24) & 0x1) == 0) && (((B >> 23) & 0x1) == 0) && (((B >> 2) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 3);

			}

			if ((((B >> 20) & 0x1) == 0) && (((B >> 17) & 0x1) == 0) && (((B >> 15) & 0x1) == 0) && (((A >> 11) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 4);

			}

			if ((((B >> 20) & 0x1) == 0) && (((B >> 9) & 0x1) == 0) && (((A >> 11) & 0x1) == 0) && (((A >> 6) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 5);

			}

			if ((((A >> 11) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 6);

			}

			if ((((B >> 10) & 0x1) == 0) && (((A >> 11) & 0x1) == 0) && (((A >> 0) & 0x1) == 0)) {
				sum1_dev[index] ^= ((((part_term_host_divide[A1]) >> (B_5)) & 0X1) << 7);

			}



		}


	}

	__syncthreads();
}


__host__ void host_creat_memory(u32** part_term) {
	u32* part_term_dev_divide;
	u32* part_term_host_divide;
	u32 B, A;
	u32 i;
	u32 rate = 64 * 2;

	u32 bolcknum = 128 * rate, threadnum = 512;
	u32 loop, offset;
	u32 t_begin, t_end;


	loop = (0x1 << 16);


	u32 bolcknum_2 = (0x1 << 11), threadnum_2 = (0x1 << 5);
	u32 sum1_size = bolcknum_2 * threadnum_2;
	u32 dim = (bolcknum * threadnum) / sum1_size;


	u32* sum1_host;
	u32* sum2_host;
	u32* sum1_dev;
	u32* sum2_dev;

	sum1_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum1_dev, sizeof(u32*) * sum1_size);


	sum2_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum2_dev, sizeof(u32*) * sum1_size);

	part_term_host_divide = (u32*)malloc(sizeof(u32) * bolcknum * threadnum);
	hipMalloc((void**)&part_term_dev_divide, sizeof(u32) * bolcknum * threadnum);

	









	//loop 16  
	for (B = 0;B < loop;B++) {

		//b   8
		//if (B == 16*16) {
		//	/*t_end = clock();
		//	printf("\n \n");
		//	printf("b: %u\n", t_begin);
		//	printf("e: %u\n", t_end);

		//	printf("b-e: %u\n", (t_end - t_begin)/ CLOCKS_PER_SEC);*/

		//	printf("B:%u\n", B);
		//	time_t end = time(nullptr);
		//	printf("time: %ld\n", (end - now));
		//	int enow = (end - now);
		//	float alltime = enow * 18.2/B;
		//	printf("maybetime:%f\n\n", (alltime));
		//	exit(0);
		//}

		global_multi_thread << <bolcknum, threadnum >> > (part_term_dev_divide, B);


		hipMemcpy(part_term_host_divide, part_term_dev_divide, sizeof(u32) * bolcknum * threadnum, hipMemcpyDeviceToHost);


		getsum << < bolcknum_2, threadnum_2 >> > (part_term_dev_divide, B, sum1_dev, dim);

		getsum2 << < bolcknum_2, threadnum_2 >> > (part_term_dev_divide, B, sum2_dev, dim);

		hipMemcpy(sum1_host, sum1_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);

		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][0] ^= sum1_host[A];
		}

		hipMemcpy(sum2_host, sum2_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);



		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][1] ^= sum2_host[A];

		}


	}

	//printf("allresult: %d \n", part_term[0][0]);

	hipFree(part_term_dev_divide);
	free(part_term_host_divide);
	hipFree(sum1_dev);
	free(sum1_host);
	hipFree(sum2_dev);
	free(sum2_host);
}


u32 cpu_prepare() {
	u32** part_term;
	u32 part_num = 40;
	part_term = (u32**)malloc(sizeof(u32*));
	for (int i = 0;i < 1;i++)
		part_term[i] = (u32*)malloc(part_num * sizeof(u32));

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < part_num;b++)
		{
			part_term[a][b] = 0;
		}
	}
	host_creat_memory(part_term);


	u32 sum = 0;

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < 2;b++)
		{
			printf("equ-num %u:  %u \n", b, part_term[a][b]);
		}
	}


	for (int a = 0;a < 32;a++)
	{
		printf("%d", (part_term[0][0] >> a) & 0x1);
	}
	printf("\n");
	for (int a = 0;a < 8;a++)
	{
		printf("%d", (part_term[0][1] >> a) & 0x1);
	}





	return 0;
}

int main(int argc, char** argv)
{
	u32 c;
	time_t now = time(nullptr);
	cpu_prepare();

	time_t end = time(nullptr);
	printf("\ntime: %ld second\n", (end - now));

	return 0;

}

