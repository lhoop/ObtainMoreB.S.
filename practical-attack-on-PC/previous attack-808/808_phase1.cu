#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <chrono>

using namespace std;
#define TYPE unsigned int
typedef signed char u8;
#define BASE_CUDA_CHECK(condition) { GPUAssert((condition), __FILE__, __LINE__); }
typedef unsigned int u32;



__device__ void Trivium_device(u32* keystream, u32* iv, u32 key_index)
{

	u32 roundnum = 808;

	//the original secret variables  
	u32 key[10] = { 41,35,190,132,225,108,214,174,82,144 };

	u32 z1 = 0;
	u32 roundnum_word = 0;
	u32 roundnum_bit = 0;
	u32 t1, t2, t3, i = 0;
	u32 s0, s1, s2, s3, s4, s5, s6, s7, s8, s9 = 0;
	u32 temp1, temp2, temp3, temp4, temp5, temp6 = 0;
	u32 z = 0;

	s0 = key[0] ^ (key[1] << 8) ^ (key[2] << 16) ^ (key[3] << 24);  //32k


	s1 = key[4] ^ (key[5] << 8) ^ (key[6] << 16) ^ (key[7] << 24);  //32k


	s2 = key[8] ^ (key[9] << 8);


	s3 = iv[0];

	s4 = iv[1];

	s5 = iv[2];


	s6 = 0;


	s7 = 0;


	s8 = 0;


	s9 = 0x00007000;  //7000  =  0111 + 12*0



	//65         92               161          176        242           287
	//s2[30]     s2[3]           s5[27]       s5[12]      s8[30]       s9[17]



	//64         91               160          175        241           286
	//s2[31]     s2[4]           s5[28]       s5[13]      s8[31]       s9[18]




	roundnum_word = roundnum / 32;
	roundnum_bit = roundnum % 32;
	for (i = 0;i < roundnum_word;i++)
	{

		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;//((s2<<30)|(s1>>2))^((s2<<3)|(s1>>29))^(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);

		t2 = temp3 ^ temp4;//((s5<<27)|(s4>>5))^((s5<<12)|(s4>>20))^((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19))^((s8<<9)|(s7>>23));

		t3 = temp5 ^ temp6;//((s8<<30)|(s7>>2))^((s9<<17)| (s8>>15))^

		//z = t1^t2^t3;

	//	t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);//(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//	t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);//(((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19)))^((s8<<9)|(s7>>23));
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//	t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);//(()&())^()
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;




		// update register 1
		s2 = (s1) & (0x1FFFFFFF);  //29��
		s1 = s0;
		s0 = t3;

		//	update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//	update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}
	if (roundnum_bit != 0)
	{
		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;
		t2 = temp3 ^ temp4;
		t3 = temp5 ^ temp6;
		z1 = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;
		//z1=z;
		//t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;

		// update register 1
		s2 = (s1) & (0x1FFFFFFF);
		s1 = s0;
		s0 = t3;

		//update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}

	temp1 = (s2 << 30) | (s1 >> 2);
	temp2 = (s2 << 3) | (s1 >> 29);
	temp3 = (s5 << 27) | (s4 >> 5);
	temp4 = (s5 << 12) | (s4 >> 20);
	temp5 = (s8 << 30) | (s7 >> 2);
	temp6 = (s9 << 17) | (s8 >> 15);



	int outbit = 0;
	z = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;

	if (roundnum_bit != 0)
		outbit = (z1 >> (31 - roundnum_bit)) & 0x1;
	else
		outbit = z >> 31;

	keystream[0] = outbit;
}

__global__ void global_multi_thread(u32* part_term_dev_divide, u32 loop) {
	//u32 loadkey[10] = { 41,35,190,132,225,108,214,174,82,144 };

	u32 tid_16, i;
	u32 keystream[1];
	u32 offset_28;
	u32 streambit = 0;
	u32 tmp;

	//0-2^16
	//0, 2, 4, 6, 8, 10, 11, 12, 14, 15, 17, 19, 20, 21, 22, 23
	tid_16 = loop;
	tmp = (threadIdx.x + blockIdx.x * blockDim.x);




	for (i = 0; i < 32; i++) {
		offset_28 = ((tmp << 5) | i);


		u32 cube[3] = { 0 };


		cube[0] = (tid_16 & 0x1) | (((tid_16 >> 1) & 0x1) << 2) | (((tid_16 >> 2) & 0x1) << 4) | (((tid_16 >> 3) & 0x1) << 6) | (((tid_16 >> 4) & 0x1) << 8) | (((tid_16 >> 5) & 0x1) << 10) | (((tid_16 >> 6) & 0x1) << 11) | (((tid_16 >> 7) & 0x1) << 12) | (((tid_16 >> 8) & 0x1) << 14) | (((tid_16 >> 9) & 0x1) << 15) | (((tid_16 >> 10) & 0x1) << 16) | (((tid_16 >> 11) & 0x1) << 18) | (((tid_16 >> 12) & 0x1) << 19) | (((tid_16 >> 13) & 0x1) << 20) | (((tid_16 >> 14) & 0x1) << 21) | (((tid_16 >> 15) & 0x1) << 22) | (((offset_28 >> 0) & 0x1) << 23) | (((offset_28 >> 1) & 0x1) << 25) | (((offset_28 >> 2) & 0x1) << 27) | (((offset_28 >> 3) & 0x1) << 29) | (((offset_28 >> 4) & 0x1) << 30);
		cube[1] = (((offset_28 >> 5) & 0x1) << 0) | (((offset_28 >> 6) & 0x1) << 2) | (((offset_28 >> 7) & 0x1) << 4) | (((offset_28 >> 8) & 0x1) << 5) | (((offset_28 >> 9) & 0x1) << 7) | (((offset_28 >> 10) & 0x1) << 9) | (((offset_28 >> 11) & 0x1) << 11) | (((offset_28 >> 12) & 0x1) << 13) | (((offset_28 >> 13) & 0x1) << 15) | (((offset_28 >> 14) & 0x1) << 18) | (((offset_28 >> 15) & 0x1) << 21) | (((offset_28 >> 16) & 0x1) << 22) | (((offset_28 >> 17) & 0x1) << 23) | (((offset_28 >> 18) & 0x1) << 25) | (((offset_28 >> 19) & 0x1) << 28) | (((offset_28 >> 20) & 0x1) << 30);
		cube[2] = (((offset_28 >> 21) & 0x1) << 0) | (((offset_28 >> 22) & 0x1) << 5) | (((offset_28 >> 23) & 0x1) << 7) | (((offset_28 >> 24) & 0x1) << 8) | (((offset_28 >> 25) & 0x1) << 11) | (((offset_28 >> 26) & 0x1) << 12) | (((offset_28 >> 27) & 0x1) << 15);

		Trivium_device(keystream, cube, i);
		streambit |= ((keystream[0] & 0x01) << (i));



	}

	/*printf("%u", streambit);*/

	part_term_dev_divide[threadIdx.x + blockIdx.x * blockDim.x] = streambit;



	__syncthreads();
}


__global__ void  getsum(u32* part_term_host_divide, u32 loop, u32* sum1_dev, u32 dim) {
	u32 tid_16 = 0;
	u32 A = 0;
	u32 offset_28 = 0;
	u32 i = 0;
	u32 tid = threadIdx.x + blockIdx.x * blockDim.x;

	sum1_dev[tid] = 0;

	for (A = dim * tid; A < dim * (tid + 1); A++) {

		for (i = 0; i < 32; i++) {

			offset_28 = ((A << 5) | i);
			tid_16 = loop;
			if ((((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0) && (((offset_28 >> 25) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 0);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 1);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 2);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 10) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 3);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 4);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 5);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 6);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((tid_16 >> 8) & 0x1) == 0) && (((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 7);
			}
			if ((((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 8);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 9);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 10);
			}
			if ((((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 15) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 11);
			}
			if ((((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 9) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 12);
			}
			if ((((tid_16 >> 9) & 0x1) == 0) && (((tid_16 >> 12) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 13);
			}
			if ((((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 14);
			}
			if ((((offset_28 >> 4) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 15);
			}
			if ((((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 16);
			}
			if ((((offset_28 >> 11) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0) && (((offset_28 >> 25) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 17);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 18);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 17) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 19);
			}
			if ((((tid_16 >> 9) & 0x1) == 0) && (((tid_16 >> 13) & 0x1) == 0) && (((offset_28 >> 25) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 20);
			}
			if ((((offset_28 >> 3) & 0x1) == 0) && (((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 21);
			}
			if ((((offset_28 >> 13) & 0x1) == 0) && (((offset_28 >> 14) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 22);
			}
			if ((((offset_28 >> 6) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0) && (((offset_28 >> 25) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 23);
			}
			if ((((tid_16 >> 9) & 0x1) == 0) && (((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 13) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 24);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 3) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 25);
			}
			if ((((offset_28 >> 5) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 26);
			}
			if ((((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 9) & 0x1) == 0) && (((offset_28 >> 13) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 27);
			}
			if ((((tid_16 >> 8) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 28);
			}
			if ((((tid_16 >> 7) & 0x1) == 0) && (((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 10) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 29);
			}
			if ((((tid_16 >> 7) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 16) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 30);
			}
			if ((((offset_28 >> 3) & 0x1) == 0) && (((offset_28 >> 21) & 0x1) == 0) && (((offset_28 >> 23) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 31);
			}






		}


	}

	__syncthreads();
}

__global__ void  getsum2(u32* part_term_host_divide, u32 loop, u32* sum1_dev, u32 dim) {
	u32 A = 0;
	u32 A1 = 0;
	u32 offset_28 = 0;
	u32 i = 0;
	u32 tid_16 = 0;
	u32 tid = threadIdx.x + blockIdx.x * blockDim.x;

	sum1_dev[tid] = 0;
	for (A = dim * tid; A < dim * (tid + 1); A++) {



		for (i = 0; i < 32;i++) {

			offset_28 = ((A << 5) | i);

			tid_16 = loop;

			if ((((tid_16 >> 9) & 0x1) == 0) && (((tid_16 >> 11) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 18) & 0x1) == 0) && (((offset_28 >> 26) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 0);
			}
			if ((((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0) && (((offset_28 >> 21) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 1);
			}
			if ((((tid_16 >> 2) & 0x1) == 0) && (((tid_16 >> 6) & 0x1) == 0) && (((offset_28 >> 8) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 2);
			}
			if ((((tid_16 >> 10) & 0x1) == 0) && (((offset_28 >> 15) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 3);
			}
			if ((((tid_16 >> 1) & 0x1) == 0) && (((offset_28 >> 24) & 0x1) == 0)) {
				sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) << 4);
			}








		}


	}

	__syncthreads();
}


__host__ void host_creat_memory(u32** part_term) {
	u32* part_term_dev_divide;
	u32* part_term_host_divide;
	u32 B, A;
	u32 i;
	u32 rate = 64 * 2;

	u32 bolcknum = 128 * rate, threadnum = 512;
	u32 loop, offset;
	u32 t_begin, t_end;


	loop = (0x1 << 16);


	u32 bolcknum_2 = (0x1 << 11), threadnum_2 = (0x1 << 5);
	u32 sum1_size = bolcknum_2 * threadnum_2;
	u32 dim = (bolcknum * threadnum) / sum1_size;


	u32* sum1_host;
	u32* sum2_host;
	u32* sum1_dev;
	u32* sum2_dev;

	sum1_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum1_dev, sizeof(u32*) * sum1_size);


	sum2_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum2_dev, sizeof(u32*) * sum1_size);

	part_term_host_divide = (u32*)malloc(sizeof(u32) * bolcknum * threadnum);
	hipMalloc((void**)&part_term_dev_divide, sizeof(u32) * bolcknum * threadnum);











	//loop 16  
	for (B = 0;B < loop;B++) {


		global_multi_thread << <bolcknum, threadnum >> > (part_term_dev_divide, B);


		hipMemcpy(part_term_host_divide, part_term_dev_divide, sizeof(u32) * bolcknum * threadnum, hipMemcpyDeviceToHost);


		getsum << < bolcknum_2, threadnum_2 >> > (part_term_dev_divide, B, sum1_dev, dim);

		getsum2 << < bolcknum_2, threadnum_2 >> > (part_term_dev_divide, B, sum2_dev, dim);

		hipMemcpy(sum1_host, sum1_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);

		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][0] ^= sum1_host[A];
		}

		hipMemcpy(sum2_host, sum2_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);



		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][1] ^= sum2_host[A];

		}


	}

	//printf("allresult: %d \n", part_term[0][0]);

	hipFree(part_term_dev_divide);
	free(part_term_host_divide);
	hipFree(sum1_dev);
	free(sum1_host);
	hipFree(sum2_dev);
	free(sum2_host);
}


u32 cpu_prepare() {
	u32** part_term;
	u32 part_num = 40;
	part_term = (u32**)malloc(sizeof(u32*));
	for (int i = 0;i < 1;i++)
		part_term[i] = (u32*)malloc(part_num * sizeof(u32));

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < part_num;b++)
		{
			part_term[a][b] = 0;
		}
	}
	host_creat_memory(part_term);


	u32 sum = 0;

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < 2;b++)
		{
			printf("equ-num %u:  %u \n", b, part_term[a][b]);
		}
	}


	for (int a = 0;a < 32;a++)
	{
		printf("%d", (part_term[0][0] >> a) & 0x1);
	}
	printf("\n");
	for (int a = 0;a < 5;a++)
	{
		printf("%d", (part_term[0][1] >> a) & 0x1);
	}





	return 0;
}

int main(int argc, char** argv)
{
	u32 c;
	time_t now = time(nullptr);
	cpu_prepare();

	time_t end = time(nullptr);
	printf("\ntime: %ld second\n", (end - now));

	return 0;

}

