#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

#include <ctime>
#include <chrono>

using namespace std;
#define TYPE unsigned int
typedef signed char u8;
#define BASE_CUDA_CHECK(condition) { GPUAssert((condition), __FILE__, __LINE__); }
typedef unsigned int u32;



__device__ void Trivium_device(u32* keystream, u32* iv, u32 key_index)
{

	u32 roundnum = 825;

	//the original secret variables  	
	u32 key[10] = { 41,35,190,132,225,108,214,174,82,144 };

	u32 z1 = 0;
	u32 roundnum_word = 0;
	u32 roundnum_bit = 0;
	u32 t1, t2, t3, i = 0;
	u32 s0, s1, s2, s3, s4, s5, s6, s7, s8, s9 = 0;
	u32 temp1, temp2, temp3, temp4, temp5, temp6 = 0;
	u32 z = 0;

	s0 = key[0] ^ (key[1] << 8) ^ (key[2] << 16) ^ (key[3] << 24);  //32k
	//           key[0]  S31-S24       key[1]   S23-S16   key[2] S15-S8  key[3]S7-S0


	s1 = key[4] ^ (key[5] << 8) ^ (key[6] << 16) ^ (key[7] << 24);  //32k


	s2 = key[8] ^ (key[9] << 8);  

	s3 = iv[0];

	s4 = iv[1];

	s5 = iv[2];


	s6 = 0;


	s7 = 0;


	s8 = 0;


	s9 = 0x00007000;  //7000  =  0111 + 12*0

	// 32*36 = 1152

	//65         92               161          176        242           287
	//s2[30]     s2[3]           s5[27]       s5[12]      s8[30]       s9[17]



	//64         91               160          175        241           286
	//s2[31]     s2[4]           s5[28]       s5[13]      s8[31]       s9[18]



	roundnum_word = roundnum / 32;
	roundnum_bit = roundnum % 32;
	for (i = 0;i < roundnum_word;i++)
	{

		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;//((s2<<30)|(s1>>2))^((s2<<3)|(s1>>29))^(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);

		t2 = temp3 ^ temp4;//((s5<<27)|(s4>>5))^((s5<<12)|(s4>>20))^((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19))^((s8<<9)|(s7>>23));

		t3 = temp5 ^ temp6;//((s8<<30)|(s7>>2))^((s9<<17)| (s8>>15))^

		//z = t1^t2^t3;


	//	t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);//(((s2<<5)|(s1>>27))&((s2<<4)|(s1>>28)))^(s5<<18)|(s4>>14);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//	t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);//(((s5<<14)|(s4>>18))&((s5<<13)|(s4>>19)))^((s8<<9)|(s7>>23));
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//	t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);//(()&())^()
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;





		// update register 1
		s2 = (s1) & (0x1FFFFFFF);  //29λ
		s1 = s0;
		s0 = t3;

		//	update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//	update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}
	if (roundnum_bit != 0)
	{
		temp1 = (s2 << 30) | (s1 >> 2);
		temp2 = (s2 << 3) | (s1 >> 29);
		temp3 = (s5 << 27) | (s4 >> 5);
		temp4 = (s5 << 12) | (s4 >> 20);
		temp5 = (s8 << 30) | (s7 >> 2);
		temp6 = (s9 << 17) | (s8 >> 15);

		t1 = temp1 ^ temp2;
		t2 = temp3 ^ temp4;
		t3 = temp5 ^ temp6;
		z1 = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;
		//z1=z;

		//t1 = t1 + s91s92 + s171
		temp1 = (s2 << 5) | (s1 >> 27);
		temp2 = (s2 << 4) | (s1 >> 28);
		temp3 = (s5 << 18) | (s4 >> 14);

		t1 ^= (temp1 & temp2) ^ temp3;

		//t2 = t2 + s175s176 + s264
		temp1 = (s5 << 14) | (s4 >> 18);
		temp2 = (s5 << 13) | (s4 >> 19);
		temp3 = (s8 << 9) | (s7 >> 23);

		t2 ^= (temp1 & temp2) ^ temp3;

		//t3 = t3 + s286s287 + s69
		temp1 = (s9 << 19) | (s8 >> 13);
		temp2 = (s9 << 18) | (s8 >> 14);
		temp3 = (s2 << 27) | (s1 >> 5);

		t3 ^= (temp1 & temp2) ^ temp3;

		// update register 1
		s2 = (s1) & (0x1FFFFFFF);
		s1 = s0;
		s0 = t3;

		//update register 2
		s5 = s4 & (0x000FFFFF);
		s4 = s3;
		s3 = t1;

		//update register 3
		s9 = s8 & (0x00007FFF);
		s8 = s7;
		s7 = s6;
		s6 = t2;
	}


	temp1 = (s2 << 30) | (s1 >> 2);
	temp2 = (s2 << 3) | (s1 >> 29);
	temp3 = (s5 << 27) | (s4 >> 5);
	temp4 = (s5 << 12) | (s4 >> 20);
	temp5 = (s8 << 30) | (s7 >> 2);
	temp6 = (s9 << 17) | (s8 >> 15);



	int outbit = 0;
	z = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;

	if (roundnum_bit != 0)
		outbit = (z1 >> (31 - roundnum_bit)) & 0x1;
	else
		outbit = z >> 31;

	keystream[0] = outbit;
	
}

__global__ void global_multi_thread(u32* part_term_dev_divide, u32 loop25) {
	//u32 loadkey[10] = { 41,35,190,132,225,108,214,174,82,144 };
	u32 tid_23, i;
	u32 keystream[1];
	u32 offset_28;
	u32 streambit = 0;
	//0-2^16
	//0, 2, 4, 6, 8, 10, 11, 12, 14, 15, 17, 19, 20, 21, 22, 23
	tid_23 = (threadIdx.x + blockIdx.x * blockDim.x);




	for (i = 0; i < 32; i++) {
		offset_28 = ((tid_23 << 5) | i);


		u32 cube[3] = { 0 };

		cube[0] = (offset_28 & 0x1) | (((offset_28 >> 1) & 0x1) << 1) | (((offset_28 >> 2) & 0x1) << 2) | (((offset_28 >> 3) & 0x1) << 3) | (((offset_28 >> 4) & 0x1) << 4) | (((offset_28 >> 5) & 0x1) << 5) | (((offset_28 >> 6) & 0x1) << 6) | (((offset_28 >> 7) & 0x1) << 7) | (((offset_28 >> 8) & 0x1) << 8) | (((offset_28 >> 9) & 0x1) << 9) | (((offset_28 >> 10) & 0x1) << 10) | (((offset_28 >> 11) & 0x1) << 12) | (((offset_28 >> 12) & 0x1) << 13) | (((offset_28 >> 13) & 0x1) << 14) | (((offset_28 >> 14) & 0x1) << 15) | (((offset_28 >> 15) & 0x1) << 16) | (((offset_28 >> 16) & 0x1) << 17) | (((offset_28 >> 17) & 0x1) << 18) | (((offset_28 >> 18) & 0x1) << 19) | (((offset_28 >> 19) & 0x1) << 21) | (((offset_28 >> 20) & 0x1) << 22) | (((offset_28 >> 21) & 0x1) << 23) | (((offset_28 >> 22) & 0x1) << 25) | (((offset_28 >> 23) & 0x1) << 26) | (((offset_28 >> 24) & 0x1) << 29) | (((offset_28 >> 25) & 0x1) << 31);
		cube[1] = (((offset_28 >> 26) & 0x1) << 1) | (((offset_28 >> 27) & 0x1) << 2) | (((loop25) & 0x1) << 3) | (((loop25 >> 1) & 0x1) << 4) | (((loop25 >> 2) & 0x1) << 5) | (((loop25 >> 3) & 0x1) << 6) | (((loop25 >> 4) & 0x1) << 7) | (((loop25 >> 5) & 0x1) << 8) | (((loop25 >> 6) & 0x1) << 9) | (((loop25 >> 7) & 0x1) << 12) | (((loop25 >> 8) & 0x1) << 14) | (((loop25 >> 9) & 0x1) << 16) | (((loop25 >> 10) & 0x1) << 18) | (((loop25 >> 11) & 0x1) << 19) | (((loop25 >> 12) & 0x1) << 21) | (((loop25 >> 13) & 0x1) << 23) | (((loop25 >> 14) & 0x1) << 25) | (((loop25 >> 15) & 0x1) << 27) | (((loop25 >> 16) & 0x1) << 29) | (((loop25 >> 17) & 0x1) << 31);
		cube[2] = (((loop25 >> 18) & 0x1) << 2) | (((loop25 >> 19) & 0x1) << 4) | (((loop25 >> 20) & 0x1) << 5) | (((loop25 >> 21) & 0x1) << 8) | (((loop25 >> 22) & 0x1) << 10) | (((loop25 >> 23) & 0x1) << 12) | (((loop25 >> 24) & 0x1) << 14);

		Trivium_device(keystream, cube, i);
		streambit |= ((keystream[0] & 0x01) << (i));

	}

	/*printf("%u", streambit);*/

	part_term_dev_divide[threadIdx.x + blockIdx.x * blockDim.x] = streambit;



	__syncthreads();
}


__global__ void  getsum(u32* part_term_host_divide, u32 loop25, u32* sum1_dev, u32* sum2_dev, u32 dim) {
	u32 offset_28 = 0;
	u32 tid = 0; 
	u32 A=0;
	u32 i=0;
	tid = threadIdx.x + blockIdx.x * blockDim.x;
	sum1_dev[tid] = 0;
	sum2_dev[tid] = 0;
	for (A = dim * tid; A < dim * (tid + 1); A++) {
		for (i = 0; i < 32; i++) {

			offset_28 = ((A << 5) | i);

			if( ((( offset_28 >> 17)&0x1)==0)  && ((( offset_28 >> 0)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<0 );
			}
			if( ((( loop25 >> 6)&0x1)==0)  && ((( loop25 >> 0)&0x1)==0)  && ((( offset_28 >> 0)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<1 );
			}
			if( ((( loop25 >> 0)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  && ((( offset_28 >> 7)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<2 );
			}
			if( ((( loop25 >> 10)&0x1)==0)  && ((( loop25 >> 0)&0x1)==0)  && ((( offset_28 >> 22)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<3 );
			}
			if( ((( loop25 >> 20)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<4 );
			}
			if( ((( loop25 >> 3)&0x1)==0)  && ((( offset_28 >> 1)&0x1)==0)  && ((( offset_28 >> 0)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<5 );
			}
			if( ((( offset_28 >> 2)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<6 );
			}
			if( ((( loop25 >> 1)&0x1)==0)  && ((( offset_28 >> 27)&0x1)==0)  && ((( offset_28 >> 21)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<7 );
			}
			if( ((( loop25 >> 20)&0x1)==0)  && ((( offset_28 >> 22)&0x1)==0)  && ((( offset_28 >> 20)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<8 );
			}
			if( ((( offset_28 >> 20)&0x1)==0)  && ((( offset_28 >> 19)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<9 );
			}
			if( ((( offset_28 >> 4)&0x1)==0)  && ((( loop25 >> 11)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<10 );
			}
			if( ((( offset_28 >> 21)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<11 );
			}
			if( ((( offset_28 >> 22)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<12 );
			}
			if( ((( offset_28 >> 16)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  && ((( offset_28 >> 9)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<13 );
			}
			if( ((( loop25 >> 17)&0x1)==0)  && ((( offset_28 >> 20)&0x1)==0)  && ((( offset_28 >> 7)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<14 );
			}
			if( ((( loop25 >> 20)&0x1)==0)  && ((( loop25 >> 17)&0x1)==0)  && ((( offset_28 >> 26)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<15 );
			}
			if( ((( loop25 >> 6)&0x1)==0)  && ((( offset_28 >> 23)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<16 );
			}
			if( ((( offset_28 >> 9)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<17 );
			}
			if( ((( offset_28 >> 23)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<18 );
			}
			if( ((( offset_28 >> 14)&0x1)==0)  && ((( offset_28 >> 10)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<19 );
			}
			if( ((( loop25 >> 17)&0x1)==0)  && ((( loop25 >> 3)&0x1)==0)  && ((( offset_28 >> 27)&0x1)==0)  && ((( offset_28 >> 22)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<20 );
			}
			if( ((( offset_28 >> 20)&0x1)==0)  && ((( offset_28 >> 18)&0x1)==0)  && ((( offset_28 >> 0)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<21 );
			}
			if( ((( loop25 >> 4)&0x1)==0)  && ((( offset_28 >> 18)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<22 );
			}
			if( ((( offset_28 >> 12)&0x1)==0)  && ((( offset_28 >> 18)&0x1)==0)  && ((( offset_28 >> 8)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<23 );
			}
			if( ((( offset_28 >> 20)&0x1)==0)  && ((( offset_28 >> 18)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<24 );
			}
			if( ((( loop25 >> 18)&0x1)==0)  && ((( loop25 >> 4)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<25 );
			}
			if( ((( loop25 >> 12)&0x1)==0)  && ((( loop25 >> 11)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  && ((( offset_28 >> 1)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<26 );
			}
			if( ((( loop25 >> 11)&0x1)==0)  && ((( offset_28 >> 26)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  && ((( offset_28 >> 4)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<27 );
			}
			if( ((( loop25 >> 3)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  && ((( offset_28 >> 1)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<28 );
			}
			if( ((( loop25 >> 20)&0x1)==0)  && ((( loop25 >> 17)&0x1)==0)  && ((( offset_28 >> 22)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<29 );
			}
			if( ((( offset_28 >> 27)&0x1)==0)  && ((( offset_28 >> 3)&0x1)==0)  && ((( offset_28 >> 0)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<30 );
			}
			if( ((( loop25 >> 4)&0x1)==0)  && ((( offset_28 >> 14)&0x1)==0)  && ((( offset_28 >> 12)&0x1)==0)  ){
			sum1_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) <<31 );
			}

			if(  ((( offset_28 >> 26)&0x1)==0)  && ((( offset_28 >> 23)&0x1)==0)  && ((( offset_28 >> 16)&0x1)==0)  ){
			sum2_dev[tid] ^= ((((part_term_host_divide[A]) >> (i)) & 0X1) );
			}


		}


	}

	__syncthreads();
}





__host__ void host_creat_memory(u32** part_term) {
	u32* part_term_dev_divide;
	u32* part_term_host_divide;
	u32 B, A;
	u32 i;
	u32 rate = 64 * 2;

	u32 bolcknum = 128 * rate, threadnum = 512;
	u32 loop, offset;
	u32 t_begin, t_end;

	loop = (0x1 << 25);

	u32 bolcknum_2 = (0x1 << 10), threadnum_2 = (0x1 << 7);
	u32 sum1_size = bolcknum_2 * threadnum_2;
	u32 dim = (bolcknum * threadnum) / sum1_size;


	u32* sum1_host;
	u32* sum2_host;
	u32* sum1_dev;
	u32* sum2_dev;;

	sum1_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum1_dev, sizeof(u32*) * sum1_size);

	sum2_host = (u32*)malloc(sizeof(u32) * sum1_size);
	hipMalloc((void**)&sum2_dev, sizeof(u32*) * sum1_size);

	part_term_host_divide = (u32*)malloc(sizeof(u32) * bolcknum * threadnum);
	hipMalloc((void**)&part_term_dev_divide, sizeof(u32) * bolcknum * threadnum);


	//printf("bolcknum*threadnum: %u*%u , sum1_size: %u, dim: %u\n", bolcknum, threadnum, sum1_size, dim);

	//64*1024*128 = 51s               9s
	//64*1024*128  sum1 4*32|16 * 32*4      ?s     33s
	//64*1024*128  sum1 4*32*4|16 * 32/4      ?s     3s
	//64*1024*128  sum1 4*32*4 *4|16 * 32/4 /4     ?s     1s(28/32)
	//64*1024*128  sum1 4*32*4 *4*4|16 * 32/4 /4 /4    ?s     1s(20/32)
	//64*1024*128  sum1 4*32*4 *4*4*4|16 * 32/4 /4 /4/4    ?s     1s(20/32)
	//64*1024*128  去除sum1   13s     2s
	
	//loop 16  
	time_t now = time(nullptr);
	for (B = 0;B < loop;B++) {
		global_multi_thread << <bolcknum, threadnum >> > (part_term_dev_divide, B);

		hipMemcpy(part_term_host_divide, part_term_dev_divide, sizeof(u32) * bolcknum * threadnum, hipMemcpyDeviceToHost);


		getsum << < bolcknum_2, threadnum_2 >> > (part_term_dev_divide, B, sum1_dev, sum2_dev, dim);
		hipMemcpy(sum1_host, sum1_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);
		hipMemcpy(sum2_host, sum2_dev, sum1_size * sizeof(u32), hipMemcpyDeviceToHost);

		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][0] ^= sum1_host[A];
			//printf("%u\n", part_term[0][0]);
		}

		

		for (A = 0;A < sum1_size;A++)
		{
			part_term[0][1] ^= sum2_host[A];

		}


	}
	//printf("allresult: %d \n", part_term[0][0]);
	hipFree(part_term_dev_divide);
	free(part_term_host_divide);
	hipFree(sum1_dev);
	free(sum1_host);
	hipFree(sum2_dev);
	free(sum2_host);
}


u32 cpu_prepare() {
	u32** part_term;
	u32 part_num = 2;
	part_term = (u32**)malloc(sizeof(u32*));
	for (int i = 0;i < 1;i++)
		part_term[i] = (u32*)malloc(part_num * sizeof(u32));

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < part_num;b++)
		{
			part_term[a][b] = 0;
		}
	}
	host_creat_memory(part_term);


	u32 sum = 0;

	for (int a = 0;a < 1;a++) {
		for (int b = 0;b < 2;b++)
		{
			printf("equ-num %u:  %u \n", b, part_term[a][b]);
		}
	}


	for (int a = 0;a < 32;a++)
	{
		printf("%d", (part_term[0][0] >> a) & 0x1);
	}
	printf("\n");

	for (int a = 0;a < 1;a++)
	{
		printf("%d", (part_term[0][1] >> a) & 0x1);
	}
	printf("\n");


	return 0;
}

int main(int argc, char** argv)
{
	u32 c;
	time_t now = time(nullptr);
	cpu_prepare();
	time_t end = time(nullptr);
	printf("time: %ld second\n", (end - now));
	return 0;
}

